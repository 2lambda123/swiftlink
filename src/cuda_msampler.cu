#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "cuda_common.h"


double TMP_LOG_ZERO = -DBL_MAX;

void founderallelegraph_print(struct gpu_state* state, int locus) {
    struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
    struct adjacent_node* tmp;
    int i, j;
    
    printf("\nFOUNDER ALLELE GRAPH:\n");
    
	for(i = 0; i < state->founderallele_count; ++i) {
        printf("%d: ", i);
        for(j = 0; j < fag->num_neighbours[i]; ++j) {
            tmp = &(fag->graph[i][j]);
            printf("{%d, %d} ", tmp->id, tmp->label);
        }
        printf("\n");
    }
    printf("\n");
}

void print_descentgraph2(struct descentgraph* dg, int ped_length, int map_length) {
    int i, j;
    
    for(i = 0; i < ped_length; ++i) {
        printf("\t%d:\t", i);
        for(j = 0; j < map_length; ++j) {
            printf( "%d%d ",
                    DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, i, j, GPU_MATERNAL_ALLELE)),
                    DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, i, j, GPU_PATERNAL_ALLELE))
            );
        }
        printf("\n");
    }
    printf("\n");
}

double tmp_log_product(double a, double b) {
    return ((a == TMP_LOG_ZERO) or (b == TMP_LOG_ZERO)) ? TMP_LOG_ZERO : a + b;
}

int get_founderallele(struct gpu_state* state, int person, int locus, int allele) {
    int current = person;
    int parent_allele = allele;
	struct person* p;
	struct descentgraph* dg = GET_DESCENTGRAPH(state);
    
    while(1) {
		p = GET_PERSON(state, current);

        if(PERSON_ISFOUNDER(p)) {
            return (current * 2) + parent_allele;
        }
        
        parent_allele = DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, current, locus, parent_allele));
                
        if(parent_allele == GPU_PATERNAL_ALLELE) {
            current = PERSON_FATHER(p);
        }
        else {
            current = PERSON_MOTHER(p);
		}
    }
}

int founderallele_add(struct founderallelegraph* fag, int mat_fa, int pat_fa, int g) {
    struct adjacent_node* tmp;
    int i;
    int neighbours;
    
    if(g == GPU_GENOTYPE_UNTYPED) {
        return 1;
    }
    
    // check to see if it exists
    // if it does, then the edge needs to have the same label
    neighbours = fag->num_neighbours[mat_fa];
    
    for(i = 0; i < neighbours; ++i) {
        tmp = &(fag->graph[mat_fa][i]);
        
        if(tmp->id == pat_fa) {
            return tmp->label == g;
        }
    }
    
    tmp = &(fag->graph[mat_fa][fag->num_neighbours[mat_fa]]);
    tmp->id = pat_fa;
    tmp->label = g;
    fag->num_neighbours[mat_fa]++;
    
    if(mat_fa != pat_fa) {
        tmp = &(fag->graph[pat_fa][fag->num_neighbours[pat_fa]]);
        tmp->id = mat_fa;
        tmp->label = g;
        fag->num_neighbours[pat_fa]++;
    }
        
    return 1;
}

int founderallelegraph_populate(struct gpu_state* state, int locus) {
	struct person* p;
	struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
	int mat;
	int pat;
	int g;
	int i;
	int legal = 1;
    
	for(i = 0; i < state->pedigree_length; ++i) {
        p = GET_PERSON(state, i);
        
        if(PERSON_ISTYPED(p)) {
            mat = get_founderallele(state, i, locus, GPU_MATERNAL_ALLELE);
            pat = get_founderallele(state, i, locus, GPU_PATERNAL_ALLELE);
            g = PERSON_GENOTYPE(p, locus);
            
            if(!founderallele_add(fag, mat, pat, g)) {
                legal = 0;
                printf("illegal!\n");
            }
		}
        
		//__syncthreads();
	}
	
	return legal;
}

double founderallelegraph_enumerate(struct gpu_state* state, int locus, int* component, int cindex) {
    int i;
    
    printf("* ");
    for(i = 0; i < cindex; ++i)
        printf("%d ", component[i]);
    printf("\n");
    
    // XXX
    
    return 1.0;
}

double founderallelegraph_likelihood(struct gpu_state* state, int locus) {
    int q[128];
    int qindex = 0;
    
    int component[128];
    int cindex;
    
    int visited[128];
    int total = state->founderallele_count;
    
    int i;
    int tmp;
    
    double tmp_prob;
    double prob = 0.0;
    
    struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
    struct adjacent_node* tmp2;
    
    
    for(i = 0; i < 128; ++i) {
        visited[i] = WHITE;
    }
    
    do {
        cindex = 0;
        
        // find start point
        for(i = 0; i < state->founderallele_count; ++i) {
            if(visited[i] == WHITE) {
                visited[i] = GREY;
                q[qindex++] = i;
                break;
            }
        }
        
        while(qindex != 0) {
            tmp = q[--qindex];
            
            for(i = 0; i < fag->num_neighbours[tmp]; ++i) {
                tmp2 = &(fag->graph[tmp][i]);
                
                if(visited[tmp2->id] == WHITE) {
                    visited[tmp2->id] = GREY;
                    q[qindex++] = tmp2->id;
                }
            }
            
            visited[tmp] = BLACK;
            total--;
            
            component[cindex++] = tmp;
        }
        
        tmp_prob = founderallelegraph_enumerate(state, locus, component, cindex);
		tmp_prob = ((tmp_prob == 0.0) ? TMP_LOG_ZERO : log(tmp_prob));
		
		prob = tmp_log_product(tmp_prob, prob);
    
    } while(total != 0);
    
    return prob;
}

void msampler_run(struct gpu_state* state, int locus) {
    struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
    int i;
    double prob;
    
    for(i = 0; i < state->founderallele_count; ++i) {
        fag->num_neighbours[i] = 0;
    }
    
    founderallelegraph_populate(state, locus);
    prob = founderallelegraph_likelihood(state, locus);
    
    printf("%f\n", prob);
    
    founderallelegraph_print(state, locus);
    print_descentgraph2(GET_DESCENTGRAPH(state), state->pedigree_length, state->map->map_length);
    
    //__syncthreads();
    
    // enumerate + likelihood
    
}

void msampler_kernel(struct gpu_state* state) {
    //int locus = blockIdx.x;
    //sampler_run(state, locus);
    
    int locus;
    
    // build fags
    for(locus = 0; locus < state->map->map_length; ++locus) {
        msampler_run(state, locus);
    }
    
    printf("\n\n");
    
    abort();
    
    // sample
    // XXX
}

void run_gpu_msampler_kernel(int numblocks, int numthreads, struct gpu_state* state) {
    //msampler_kernel<<<numblocks, numthreads>>>(state);
    msampler_kernel(state);
}

