#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "cuda_common.h"


double TMP_LOG_ZERO = -DBL_MAX;

void founderallelegraph_print(struct gpu_state* state, int locus) {
    struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
    struct adjacent_node* tmp;
    int i, j;
    
    printf("\nFOUNDER ALLELE GRAPH:\n");
    
	for(i = 0; i < state->founderallele_count; ++i) {
        printf("%d: ", i);
        for(j = 0; j < fag->num_neighbours[i]; ++j) {
            tmp = &(fag->graph[i][j]);
            printf("{%d, %d} ", tmp->id, tmp->label);
        }
        printf("\n");
    }
    printf("\n");
}

void print_descentgraph2(struct descentgraph* dg, int ped_length, int map_length) {
    int i, j;
    
    for(i = 0; i < ped_length; ++i) {
        printf("\t%d:\t", i);
        for(j = 0; j < map_length; ++j) {
            printf( "%d%d ",
                    DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, i, j, GPU_MATERNAL_ALLELE)),
                    DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, i, j, GPU_PATERNAL_ALLELE))
            );
        }
        printf("\n");
    }
    printf("\n");
}

double tmp_log_product(double a, double b) {
    return ((a == TMP_LOG_ZERO) or (b == TMP_LOG_ZERO)) ? TMP_LOG_ZERO : a + b;
}

double tmp_log_sum(double a, double b) {
    if(a == TMP_LOG_ZERO)
        return b;
    
    if(b == TMP_LOG_ZERO)
        return a;
    
    return log(exp(b - a) + 1) + a;
}

int founderallele_add(struct founderallelegraph* fag, int mat_fa, int pat_fa, int g) {
    struct adjacent_node* tmp;
    int i;
    int neighbours;
    
    if(g == GPU_GENOTYPE_UNTYPED) {
        return 1;
    }
    
    // check to see if it exists
    // if it does, then the edge needs to have the same label
    neighbours = fag->num_neighbours[mat_fa];
    
    for(i = 0; i < neighbours; ++i) {
        tmp = &(fag->graph[mat_fa][i]);
        
        if(tmp->id == pat_fa) {
            return tmp->label == g;
        }
    }
    
    tmp = &(fag->graph[mat_fa][fag->num_neighbours[mat_fa]]);
    tmp->id = pat_fa;
    tmp->label = g;
    fag->num_neighbours[mat_fa]++;
    
    if(mat_fa != pat_fa) {
        tmp = &(fag->graph[pat_fa][fag->num_neighbours[pat_fa]]);
        tmp->id = mat_fa;
        tmp->label = g;
        fag->num_neighbours[pat_fa]++;
    }
        
    return 1;
}

int founderallelegraph_populate(struct gpu_state* state, int locus) {
	struct person* p;
	struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
	struct descentgraph* dg = GET_DESCENTGRAPH(state);
	int g;
	int i;
	int legal = 1;
	int pid;
	int founderalleles[256];
	int parent_allele;
	int mat, pat;
	
	// find founder allele assignments, this is only related to the current 
	// descent graph and not whether people are typed or not
	for(i = 0; i < state->pedigree_length; ++i) {
	    pid = state->fa_sequence[i];
	    p = GET_PERSON(state, pid);
	    
	    if(PERSON_ISFOUNDER(p)) {
	        founderalleles[pid * 2] = pid * 2;
	        founderalleles[(pid * 2) + 1] = (pid * 2) + 1;
	    }
	    else {
	        parent_allele = DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, pid, locus, GPU_MATERNAL_ALLELE));
	        founderalleles[pid * 2] = founderalleles[ (PERSON_MOTHER(p) * 2) + parent_allele ];
	        
	        parent_allele = DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, pid, locus, GPU_PATERNAL_ALLELE));
	        founderalleles[(pid * 2) + 1] = founderalleles[ (PERSON_FATHER(p) * 2) + parent_allele ];
	    }
	}
	
	// construct the actual graph from the assignments and the genotype
	// information
	for(i = 0; i < state->pedigree_length; ++i) {
	    pid = state->fa_sequence[i];
	    p = GET_PERSON(state, pid);
	    
	    if(PERSON_ISTYPED(p)) {
	        mat = founderalleles[pid * 2];
	        pat = founderalleles[(pid * 2) + 1];
	        g = PERSON_GENOTYPE(p, locus);
	        
	        if(! founderallele_add(fag, mat, pat, g)) {
                legal = 0;
                printf("illegal (locus = %d, person = %d, [%d %d %d])!\n", locus, i, mat, pat, g);
            }
        }
	}
	
	return legal;
}

// for loops in the allele graph, hetero is always a contradiction
int correct_alleles_loop(int g, int allele1) {
    switch(g) {
        case GPU_GENOTYPE_AA:
            return allele1 == 1;
        case GPU_GENOTYPE_BB:
            return allele1 == 2;
        case GPU_GENOTYPE_AB:
            //return 0;
        case GPU_GENOTYPE_UNTYPED:
            //abort();
            break;
    }
    return 0;
}

int correct_alleles(int g, int allele1) {
    switch(g) {
        case GPU_GENOTYPE_AA:
            return allele1 == 1;
        case GPU_GENOTYPE_BB:
            return allele1 == 2;
        case GPU_GENOTYPE_AB:
            return 1;
        case GPU_GENOTYPE_UNTYPED:
            //abort();
            break;
    }
    return 0;
}

int correct_alleles(int g, int allele1, int allele2) {
    switch(g) {
        case GPU_GENOTYPE_AA:
            return (allele1 == 1) && (allele2 == 1);
        case GPU_GENOTYPE_BB:
            return (allele1 == 2) && (allele2 == 2);
        case GPU_GENOTYPE_AB:
            return ((allele1 == 1) && (allele2 == 2)) || \
                   ((allele1 == 2) && (allele2 == 1));
        case GPU_GENOTYPE_UNTYPED:
            //abort();
            break;
    }
    return 0;
}


int legal(struct gpu_state* state, int locus, int* component, int clength, int* q, int qlength) {
//bool FounderAlleleGraph2::legal(GraphComponent& gc, vector<unsigned>& assignment) {
    //int node = gc[assignment.size() - 1];
    //int allele = assignment.back();
    int node = component[qlength-1];
    int allele = q[qlength-1];
    int i, j;  
    
    //AdjacencyRecord& tmp = matrix[node];
    struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
    struct adjacent_node* adj;

    //for(unsigned i = 0; i < tmp.size(); ++i) {
    for(i = 0; i < fag->num_neighbours[node]; ++i) {
        //FounderAlleleNode& adj = tmp[i];
        adj = &(fag->graph[node][i]);
        
        // if there is a loop
        //if(adj.id == node) {
        //    if(not correct_alleles_loop(adj.label, allele)) {
        //        return false;
        //    }
        //    continue;
        //}
        
        // if there is a loop
        if(adj->id == node) {
            if(! correct_alleles_loop(adj->label, allele)) {
                return 0;
            }
            continue;
        }
        
        // test if compatible with label
        //if(not correct_alleles(adj.label, allele)) {
        //    return false;
        //}
        
        // test if compatible with label
        if(! correct_alleles(adj->label, allele)) {
            return 0;
        }
        
        // find offset of adjacent node in assignment vector
        //unsigned j;
        //for(j = 0; j < gc.size(); ++j) {
        //    if(gc[j] == adj.id)
        //        break;
        //}
        
        // find offset of adjacent node in assignment vector
        for(j = 0; j < clength; ++j) {
            if(component[j] == adj->id) {
                break;
            }
        }
        
        // error if not found
        //if(j == gc.size()) {
        //    fprintf(stderr, "Error: an adjacent allele in the graph was not "
        //                    "found in the same component (%s:%d)", 
        //                    __FILE__, __LINE__);
        //    abort();
        //}

        // error if not found
        if(j == clength) {
            // XXX abort on gpu?
            abort();
            //return 0;
        }
        
        // if not assigned yet, then ignore
        //if(j > (assignment.size() - 1))
        //    continue;

        // if not assigned yet, then ignore
        if(j > (qlength-1))
            continue;
        
        // if assigned, then test legality
        //if(not correct_alleles(adj.label, allele, assignment[j])) {
        //    return false;
        //}

        // if assigned, then test legality
        if(! correct_alleles(adj->label, allele, q[j])) {
            return 0;
        }

    }
    
    return 1;
}

double component_likelihood(struct gpu_state* state, int locus, int* q, int length) {
    struct geneticmap* map = GET_MAP(state);
    double minor = MAP_MINOR(map, locus);
    double major = MAP_MAJOR(map, locus);
    double tmp = 1.0;
    int i;    
    
    for(i = 0; i < length; ++i) {
        tmp *= ((q[i] == 1) ? major : minor);
    }
    
    return tmp;
}

double founderallelegraph_enumerate(struct gpu_state* state, int locus, int* component, int cindex) {
    int q[128];
    int qindex = 0;
    int skip = 0;
    double prob = 0.0;
    int i;

    // <debug>
    printf("* ");
    for(i = 0; i < cindex; ++i)
        printf("%d ", component[i]);
    printf("\n");
    // </debug>
    
    if(cindex == 1) {
        return 1.0;
    }

    while(1) {
        
        while(qindex != cindex) {
            q[qindex++] = 1; // push
            if(! legal(state, locus, component, cindex, q, qindex)) {
                skip = 1;
                break;
            }
        }
        
        if(!skip) {
            prob += component_likelihood(state, locus, q, qindex);
        }
        
        while(1) {
            
            while((qindex != 0) and (q[qindex-1] == 2)) { // not empty and last value is 2
                qindex--; // pop
            }
        
            if(qindex == 0) {
                goto no_more_assignments;
            }
        
            q[qindex-1] = 2; // set last value to 2
            
            if(legal(state, locus, component, cindex, q, qindex)) {
                skip = 0;
                break;
            }
        }
    }
       
no_more_assignments:
    
    return prob;
}

double founderallelegraph_likelihood(struct gpu_state* state, int locus) {
    int q[128];
    int qindex = 0;
    
    int component[128];
    int cindex;
    
    int visited[128];
    int total = state->founderallele_count;
    
    int i;
    int tmp;
    
    double tmp_prob;
    double prob = 0.0;
    
    struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
    struct adjacent_node* tmp2;
    
    
    for(i = 0; i < 128; ++i) {
        visited[i] = _WHITE;
    }
    
    do {
        cindex = 0;
        
        // find start point
        for(i = 0; i < state->founderallele_count; ++i) {
            if(visited[i] == _WHITE) {
                visited[i] = _GREY;
                q[qindex++] = i;
                break;
            }
        }
        
        while(qindex != 0) {
            tmp = q[--qindex];
            
            for(i = 0; i < fag->num_neighbours[tmp]; ++i) {
                tmp2 = &(fag->graph[tmp][i]);
                
                if(visited[tmp2->id] == _WHITE) {
                    visited[tmp2->id] = _GREY;
                    q[qindex++] = tmp2->id;
                }
            }
            
            visited[tmp] = _BLACK;
            total--;
            
            component[cindex++] = tmp;
        }
        
        tmp_prob = founderallelegraph_enumerate(state, locus, component, cindex);
		tmp_prob = ((tmp_prob == 0.0) ? TMP_LOG_ZERO : log(tmp_prob));
		
		prob = tmp_log_product(tmp_prob, prob);
    
    } while(total != 0);
    
    return prob;
}

int founderallele_sample(struct founderallelegraph* fag) {
    double total; // = fag->prob[0] + fag->prob[1];
    
    if((fag->prob[0] == TMP_LOG_ZERO) && (fag->prob[1] == TMP_LOG_ZERO)) {
        abort();
    }
    
    if(fag->prob[0] == TMP_LOG_ZERO)
        return 1;
        
    if(fag->prob[1] == TMP_LOG_ZERO)
        return 0;
    
    total = fag->prob[0] + fag->prob[1];
    
    return (rand() / double(RAND_MAX)) < (fag->prob[0] / total) ? 0 : 1;
}

double founderallele_run(struct gpu_state* state, int locus, int personid, int allele, int value) {
    struct founderallelegraph* fag = GET_FOUNDERALLELEGRAPH(state, locus);
    struct descentgraph* dg = GET_DESCENTGRAPH(state);
    int i;
    int tmp;
    int populate_legal;
    double prob = TMP_LOG_ZERO;
    
    // save the previous value
    i = DESCENTGRAPH_OFFSET(dg, personid, locus, allele);
    tmp = DESCENTGRAPH_GET(dg, i);
    DESCENTGRAPH_SET(dg, i, value);
    
    for(i = 0; i < state->founderallele_count; ++i) {
        fag->num_neighbours[i] = 0;
    }
    
    populate_legal = founderallelegraph_populate(state, locus);
    
    if(populate_legal) {
        prob = founderallelegraph_likelihood(state, locus);
    }
    
    //printf("%f\n", prob);
    
    // restore previous value
    i = DESCENTGRAPH_OFFSET(dg, personid, locus, allele);
    DESCENTGRAPH_SET(dg, i, tmp);
    
    return prob;
}

void msampler_kernel(struct gpu_state* state, int meiosis) {
    //int locus = blockIdx.x;
    //sampler_run(state, locus);
    
    /*
    printf("fa_sequence = ");
    for(i = 0; i < state->pedigree_length; ++i) {
        printf("%d ", state->fa_sequence[i]);
    }
    printf("\n");
    */
    
    int locus;
    int personid = (state->founderallele_count / 2) + (meiosis / 2);
    int allele = meiosis % 2;
    struct founderallelegraph* fag;
    struct geneticmap* map;
    struct descentgraph* dg = GET_DESCENTGRAPH(state);
    int i, j;
    
    map = GET_MAP(state);
        
    // build fags
    for(locus = 0; locus < map->map_length; ++locus) {
        fag = GET_FOUNDERALLELEGRAPH(state, locus);
        
        fag->prob[0] = founderallele_run(state, locus, personid, allele, 0);
        fag->prob[1] = founderallele_run(state, locus, personid, allele, 1);
    }
    
    
    // sampling
    
    // forward
    for(i = 1; i < state->map->map_length; ++i) {
        for(j = 0; j < 2; ++j) {
            state->graphs[i].prob[j] = tmp_log_product( \
                                            state->graphs[i].prob[j], \
                                            tmp_log_sum( \
                                                tmp_log_product(state->graphs[i-1].prob[j],   MAP_THETA(map, i-1)), \
                                                tmp_log_product(state->graphs[i-1].prob[1-j], MAP_INVERSETHETA(map, i-1)) \
                                            ) \
                                        );
        }
    }
    
    /*
    // print
    for(locus = 0; locus < state->map->map_length; ++locus) {
        fag = GET_FOUNDERALLELEGRAPH(state, locus);
        printf("%d %.3f %.3f\n", locus, fag->prob[0], fag->prob[1]);
    }
    */
    
    // backward
    i = map->map_length - 1;
    fag = GET_FOUNDERALLELEGRAPH(state, i);
    DESCENTGRAPH_SET(dg, DESCENTGRAPH_OFFSET(dg, personid, i, allele), founderallele_sample(fag));
    
    while(--i >= 0) {
        fag = GET_FOUNDERALLELEGRAPH(state, i);
        
        for(j = 0; j < 2; ++j) {
            fag->prob[j] = tmp_log_product(fag->prob[j], ((DESCENTGRAPH_GET(dg, DESCENTGRAPH_OFFSET(dg, personid, i+1, allele)) != j) ? \
                log(MAP_THETA(map, i)) : \
                log(MAP_INVERSETHETA(map, i))));
        }
        
        DESCENTGRAPH_SET(dg, DESCENTGRAPH_OFFSET(dg, personid, i, allele), founderallele_sample(fag));
    }
    
    /*
    // print
    for(locus = 0; locus < state->map->map_length; ++locus) {
        fag = GET_FOUNDERALLELEGRAPH(state, locus);
        printf("%d %.3f %.3f\n", locus, fag->prob[0], fag->prob[1]);
    }
    */
}

void run_gpu_msampler_kernel(int numblocks, int numthreads, struct gpu_state* state, int meiosis) {
    //msampler_kernel<<<numblocks, numthreads>>>(state, meiosis);
    msampler_kernel(state, meiosis);
}

